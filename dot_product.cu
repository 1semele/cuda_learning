#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>



__global__ void dot_product(float *A, float *B, float *C, int len) {
	float sum = 0.0;
	for (int i = 0; i < len; i++) {
		sum = A[i] * B[i];
	}
	C[0] = sum;
	
}


int main() {
	int len = 10;
	float *A = (float *) malloc(sizeof(float) * len);
	float *B = (float *) malloc(sizeof(float) * len);
	float *C = (float *) malloc(sizeof(float));

	for (int i = 0; i < len; i++) {
		A[i] = i;
		B[i] = i + 1;
	}

	float *d_A, *d_B, *d_C;

	hipMalloc((void **)&d_A, sizeof(float) * len);
	hipMalloc((void **)&d_B, sizeof(float) * len);
	hipMalloc((void **)&d_C, sizeof(float) * len);

	hipMemcpy(d_A, A, sizeof(float) * len, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float) * len, hipMemcpyHostToDevice);

	dot_product<<<1, 1>>>(d_A, d_B, d_C, len);

	hipMemcpy(C, d_C, sizeof(float) * len, hipMemcpyDeviceToHost);

	printf("%f\n", C[0]);

	return 0;
}


