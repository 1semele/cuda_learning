#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>



__global__ void vector_sum(float *A, float *B, float *C, int len) {
	for (int i = 0; i < len; i++) {
		C[i] = A[i] + B[i];
	}
}


int main() {
	int len = 10;
	float *A = (float *) malloc(sizeof(float) * len);
	float *B = (float *) malloc(sizeof(float) * len);
	float *C = (float *) malloc(sizeof(float) * len);

	for (int i = 0; i < len; i++) {
		A[i] = i;
		B[i] = i + 1;
	}

	float *d_A, *d_B, *d_C;

	hipMalloc((void **)&d_A, sizeof(float) * len);
	hipMalloc((void **)&d_B, sizeof(float) * len);
	hipMalloc((void **)&d_C, sizeof(float) * len);

	hipMemcpy(d_A, A, sizeof(float) * len, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float) * len, hipMemcpyHostToDevice);

	vector_sum<<<1, 1>>>(d_A, d_B, d_C, len);

	hipMemcpy(C, d_C, sizeof(float) * len, hipMemcpyDeviceToHost);

	for (int i = 0; i < len; i++) {
		printf("%f, ", C[i]);
	}

	printf("\n");

	return 0;
}


